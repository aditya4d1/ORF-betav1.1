#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include"hip/hip_runtime.h"
#include<Windows.h>

using namespace std;

const int Length = 9;

__global__ void Kernel(char *Strd,int *Startd,int *Stopd,char *Finald){
	int tx=threadIdx.x;
	int t=tx*3;
	__shared__ char Strs[Length];
	for(int i=0;i<3;i++){
		Strs[t+i]=Strd[t+i];
	}
	for(int j=0;j<3;j++){
		if(Strs[t+j]=='a'&&Strs[t+1+j]=='t'&&Strs[t+2+j]=='g'){
			Startd[j]=t+2;
		}
		if((Strs[t+j]=='t'&&Strs[t+1+j]=='a'&&Strs[t+2+j]=='g')||(Strs[t+j]=='t'&&Strs[t+1+j]=='a'&&Strs[t+2+j]=='a')||(Strs[t+j]=='t'&&Strs[t+1+j]=='g'&&Strs[t+2+j]=='a')){
			Stopd[j]=t;
		}
		if((Startd[j]<t)&&(Stopd[j]>=t)){
 			if(Strs[t+j]=='c'&&Strs[t+1+j]=='t'&&(Strs[t+2+j]=='c'||Strs[t+2+j]=='t'||Strs[t+2+j]=='a'||Strs[t+2+j]=='g')){
				Finald[t+j]='R';
			}
 			if(Strs[t+j]=='c'&&Strs[t+1+j]=='a'&&(Strs[t+2+j]=='a'||Strs[t+j+2]=='g')){
				Finald[t+j]='Q';
			}
 			if(Strs[t+j]=='c'&&Strs[t+j+1]=='a'&&(Strs[t+j+2]=='t'||Strs[t+j+2]=='c')){
				Finald[t+j]='H';
			}
 			if(Strs[t+j]=='c'&&Strs[t+j+1]=='c'&&(Strs[t+j+2]=='c'||Strs[t+j+2]=='t'||Strs[t+j+2]=='a'||Strs[t+j+2]=='g')){
				Finald[t+j]='P';
			}
 			if(Strs[t+j]=='c'&&Strs[t+j+1]=='u'&&(Strs[t+j+2]=='c'||Strs[t+j+2]=='t'||Strs[t+j+2]=='a'||Strs[t+j+2]=='g')){
				Finald[t+j]='L';
			}
 			if(Strs[t+j]=='t'&&Strs[t+j+1]=='g'&&Strs[t+j+2]=='g'){
				Finald[t+j]='W';
			}
 			if(Strs[t+j]=='t'&&Strs[t+j+1]=='a'&&(Strs[t+j+2]=='a'||Strs[t+j+2]=='c')){
				Finald[t+j]='C';
			}
 			if(Strs[t+j]=='t'&&Strs[t+j+1]=='a'&&(Strs[t+j+2]=='u'||Strs[t+j+2]=='c')){
				Finald[t+j]='Y';
			}
 			if(Strs[t+j]=='t'&&Strs[t+j+1]=='c'&&(Strs[t+j+2]=='c'||Strs[t+j+2]=='t'||Strs[t+j+2]=='a'||Strs[t+2]=='g')){
				Finald[t+j]='S';
			}
 			if(Strs[t+j]=='t'&&Strs[t+j+1]=='t'&&(Strs[t+j+2]=='a'||Strs[t+j+2]=='g')){
				Finald[t+j]='L';
			}
 			if(Strs[t+j]=='t'&&Strs[t+j+1]=='t'&&(Strs[t+j+2]=='u'||Strs[t+j+2]=='c')){
				Finald[t+j]='F';
			}
 			if(Strs[t+j]=='g'&&Strs[t+j+1]=='g'&&(Strs[t+j+2]=='c'||Strs[t+j+2]=='t'||Strs[t+j+2]=='a'||Strs[t+j+2]=='g')){
				Finald[t+j]='G';
			}
 			if(Strs[t+j]=='g'&&Strs[t+j+1]=='a'&&(Strs[t+j+2]=='a'||Strs[t+j+2]=='g')){
				Finald[t+j]='E';
			}
 			if(Strs[t+j]=='g'&&Strs[t+j+1]=='a'&&(Strs[t+j+2]=='u'||Strs[t+j+2]=='c')){
				Finald[t+j]='D';
			}
 			if(Strs[t+j]=='g'&&Strs[t+j+1]=='c'&&(Strs[t+j+2]=='c'||Strs[t+j+2]=='t'||Strs[t+j+2]=='a'||Strs[t+j+2]=='g')){
				Finald[t+j]='A';
			}
 			if(Strs[t+j]=='g'&&Strs[t+j+1]=='t'&&(Strs[t+j+2]=='c'||Strs[t+j+2]=='t'||Strs[t+j+2]=='a'||Strs[t+j+2]=='g')){
				Finald[t+j]='V';
			}
 			if(Strs[t+j]=='a'&&Strs[t+j+1]=='g'&&(Strs[t+j+2]=='g'||Strs[t+j+2]=='a')){
				Finald[t+j]='r';
			}
 			if(Strs[t+j]=='a'&&Strs[t+j+1]=='g'&&(Strs[t+j+2]=='c'||Strs[t+j+2]=='t')){
				Finald[t+j]='S';
			}
 			if(Strs[t+j]=='a'&&Strs[t+j+1]=='a'&&(Strs[t+j+2]=='g'||Strs[t+j+2]=='a')){
				Finald[t+j]='K';
			}
 			if(Strs[t+j]=='a'&&Strs[t+j+1]=='a'&&(Strs[t+j+2]=='c'||Strs[t+j+2]=='t')){
				Finald[t+j]='N';
			}
 			if(Strs[t+j]=='a'&&Strs[t+j+1]=='c'&&(Strs[t+j+2]=='c'||Strs[t+j+2]=='t'||Strs[t+j+2]=='a'||Strs[t+j+2]=='g')){
				Finald[t+j]='T';
			}
 			if(Strs[t+j]=='a'&&Strs[t+j+1]=='t'&&(Strs[t+j+2]=='c'||Strs[t+j+2]=='t'||Strs[t+j+2]=='a')){
				Finald[t+j]='I';
			}
		}
	}
	for(int i=0;i<3;i++){
		Strd[t+i]=Strs[t+i];
	}
//	Finald[t]='a'+t;
}

int main(){
	char Str[Length],Final[Length],*Strd,*Finald;
	int Start[3],Stop[3],*Startd,*Stopd;
	for(int i=0;i<Length;i++){
		cin>>Str[i];
		Final[i]='a'+i;
	}
	for(int i=0;i<3;i++){
		Start[i]=i;
		Stop[i]=i;
	}
	dim3 dimBlock(Length/3,1);
	dim3 dimGrid(1,1);
	hipMalloc((void**)&Strd,Length*sizeof(char));
	hipMalloc((void**)&Startd,3*sizeof(int));
	hipMalloc((void**)&Stopd,3*sizeof(int));
	hipMalloc((void**)&Finald,Length*sizeof(char));
	hipMemcpy(Strd,Str,Length*sizeof(char),hipMemcpyHostToDevice);
	Kernel<<<dimGrid,dimBlock>>>(Strd,Startd,Stopd,Finald);
	hipMemcpy(Final,Finald,Length*sizeof(char),hipMemcpyDeviceToHost);
	hipMemcpy(Start,Startd,3*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(Stop,Stopd,3*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(Str,Strd,Length*sizeof(char),hipMemcpyDeviceToHost);
	for(int i=0;i<3;i++){
		cout<<Start[i]<<"	"<<Stop[i]<<endl;
	}
	for(int i=0;i<Length;i++){
		cout<<Str[i]<<" - "<<i<<endl;
		cout<<Final[i]<<" - "<<i<<endl;
	}
	system("pause");
	return 0;
}